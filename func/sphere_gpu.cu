#include "hip/hip_runtime.h"
#include "sphere_gpu.h"

// vetor de entrada armazenado na GPU
real *d_x;

// dimensionalidade do vetor de entrada
int dim;

// funcoes utilitarias
inline real* allocateRealsDev(int n)
{
    real *res;

    if (hipMalloc((void**) &res, n * sizeof(real)) != hipSuccess) {
        return NULL;
    }

    return res;
}

// inicializa o armazenamento na GPU
bool init_gpu(int n)
{
    d_x = allocateRealsDev(n);
    if (d_x == NULL)
        return false;

    dim = n;
    return true;
}

void finalize_gpu()
{
    if (d_x != NULL) {
        hipFree(d_x);
        d_x = NULL;
    }
}


// kernels
__global__ void square_kernel(real *d_x, real *d_res)
{
    int tid = threadIdx.x;
    real x = d_x[tid];

    d_res[tid] = x * x;
}

__global__ void reduction_kernel(real *vec, int span)
{
    int tid = threadIdx.x * span;

    vec[tid] = vec[tid] + vec[tid + span/2];
}

real reduce_sum(int n, real *d_vec)
{
    real *h_vec;
    real sum = 0.0;

    h_vec = (real*) malloc(n * sizeof(real));
    if (h_vec == NULL)
        return 0.0;
    hipMemcpy(h_vec, d_vec, n * sizeof(real), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
        sum += h_vec[i];

    free(h_vec);
    return sum;
}

real sphere_gpu(real *x)
{
    real res;
    real *d_res;

    d_res = allocateRealsDev(dim);
    if (d_res == NULL)
        return -1.0f;

    // transfere vetor para GPU
    hipMemcpy(d_x, x, dim * sizeof(real), hipMemcpyHostToDevice);

    square_kernel<<<1, dim>>>(d_x, d_res);

    // reducao
    res = reduce_sum(dim, d_res);

    hipFree(d_res);

    return res;
}
